#include "hip/hip_runtime.h"
#include "wingheader.h"

#define MAX_B 1
#define MAX_THREAD 1024
#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)
#define cudaSafeCall(call)  \
        do {\
            hipError_t err = call;\
            if (hipSuccess != err) \
            {\
                std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__ << "): " \
                    << hipGetErrorString(err);\
                exit(EXIT_FAILURE);\
            }\
        } while(0)
#define ACCESS(arr, off, ind) ((arr)[(off) + (ind)])
#define ACCESS2D(arr, indx, indy) ((arr)[(indx)][(indy)])


void gpu_error(hipError_t const &code) {
    if(code != hipSuccess)
    {
        std::cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << std::endl;
        exit( EXIT_FAILURE );
    }
}

__global__ void precompute(int och, int ch, float* kernel_weights, float *U)
{
    int tch = blockIdx.x;
    int toch = threadIdx.x;
   
    float au, bu, cu, du, eu, fu, gu, hu, iu;
    int ind = 0;
    int offset = (toch*ch + tch)*9;

    au = ACCESS(kernel_weights, offset, ind++);
    bu = ACCESS(kernel_weights, offset, ind++);
    cu = ACCESS(kernel_weights, offset, ind++);
    du = ACCESS(kernel_weights, offset, ind++);
    eu = ACCESS(kernel_weights, offset, ind++);
    fu = ACCESS(kernel_weights, offset, ind++);
    gu = ACCESS(kernel_weights, offset, ind++);
    hu = ACCESS(kernel_weights, offset, ind++);
    iu = ACCESS(kernel_weights, offset, ind++);
    
    ind = 0;
    offset = (toch*ch + tch)*16;

    float adg, beh, cfi, a_dg, b_eh, c_fi;
    adg = au+du+gu;
    beh = bu+eu+hu;
    cfi = cu+fu+iu;
    a_dg = au-du+gu;
    b_eh = bu-eu+hu;
    c_fi = cu-fu+iu;
    
    ACCESS(U, offset, ind++) = au;
    ACCESS(U, offset, ind++) = 0.5*(au+bu+cu);
    ACCESS(U, offset, ind++) = 0.5*(au-bu+cu);
    ACCESS(U, offset, ind++) = cu;
    ACCESS(U, offset, ind++) = 0.5*(adg);
    ACCESS(U, offset, ind++) = 0.25*(adg+beh+cfi);
    ACCESS(U, offset, ind++) = 0.25*(adg-beh+cfi);
    ACCESS(U, offset, ind++) = 0.5*(cfi);
    ACCESS(U, offset, ind++) = 0.5*(a_dg);
    ACCESS(U, offset, ind++) = 0.25*(a_dg+b_eh+c_fi);
    ACCESS(U, offset, ind++) = 0.25*(a_dg-b_eh+c_fi);
    ACCESS(U, offset, ind++) = 0.5*(c_fi);
    ACCESS(U, offset, ind++) = gu;
    ACCESS(U, offset, ind++) = 0.5*(gu+hu+iu);
    ACCESS(U, offset, ind++) = 0.5*(gu-hu+iu);
    ACCESS(U, offset, ind++) = iu;
    
}


__global__ void paddev(float *devin, float *devinnopad, int h, int w, int pad)
{
    int newh = gridDim.y;
    int neww = gridDim.z;
    int tbs = blockIdx.x;
    int tch = threadIdx.x;
    int ch = blockDim.x;
    int tnewh = blockIdx.y;
    int tneww = blockIdx.z;
    int newhw = newh*neww;
    int hw = h*w;
    int th = tnewh-pad;
    int tw = tneww-pad;
    int tbsch = tbs*ch + tch;
    
    if(th >= 0 && th < h && tw >= 0 && tw < w)
        devin[tbsch*newhw + tnewh*neww + tneww] = devinnopad[tbsch*hw + th*w + tw];
    else
        devin[tbsch*newhw + tnewh*neww + tneww] = 0;
    
}

__global__ void cutpad(float  *devY, float *devcutY, int oph,int opw)
{
    int p = gridDim.y;
    int q = gridDim.z;
    int tbs = blockIdx.x;
    int tp = blockIdx.y;
    int tq = blockIdx.z;
    int toch = threadIdx.x;
    int och = blockDim.x;
    int offset = tbs*och+toch;
    int ophopw = oph*opw;
    for(int i = 0; i < 2; i++)
    {
        for(int j = 0; j < 2; j++)
        {
            if(tp*2 + i < oph && tq*2 + j < opw)
                devcutY[offset*ophopw + (tp*2+i)*opw + (tq*2+j)] = devY[(((offset*p + tp)*q +tq)*2 + i)*2  + j];
        }
    }
}

__global__ void tile(int bs, int p, int q, int ch, float *devin, float *devsum, float *devU, int h, int w, int och, float *devfin)
{
    int tbs, tp, tq, tch, Tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    Tch = threadIdx.x;
    tch = Tch / och;

    float V[4][4];
    
    // copy the tiles to thrtile
    // int offset1 = (tbs*ch + tch)*h*w;
    // for(int th = 2*tp, i = 0; i < 4; th++, i++)
    //     for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
    //         thrtile[i][j] = devin[offset1 + th*w + tw];

    int offset1 = (tbs*ch + tch)*h*w;
    
    float av, bv, cv, dv, ev, fv, gv, hv, iv, jv, kv, lv, mv, nv, ov, pv;
    int th = 2*tp, tw = 2*tq;
    av = ACCESS(devin, offset1, th*w + tw++);
    bv = ACCESS(devin, offset1, th*w + tw++);
    cv = ACCESS(devin, offset1, th*w + tw++);
    dv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    ev = ACCESS(devin, offset1, th*w + tw++);
    fv = ACCESS(devin, offset1, th*w + tw++);
    gv = ACCESS(devin, offset1, th*w + tw++);
    hv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    iv = ACCESS(devin, offset1, th*w + tw++);
    jv = ACCESS(devin, offset1, th*w + tw++);
    kv = ACCESS(devin, offset1, th*w + tw++);
    lv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    mv = ACCESS(devin, offset1, th*w + tw++);
    nv = ACCESS(devin, offset1, th*w + tw++);
    ov = ACCESS(devin, offset1, th*w + tw++);
    pv = ACCESS(devin, offset1, th*w + tw++);
    
    //Calculation of V
    int vx = 0, vy = 0;
    ACCESS2D(V, vx, vy++) = +av-iv-cv+kv;
    ACCESS2D(V, vx, vy++) = +bv-jv+cv-kv;
    ACCESS2D(V, vx, vy++) = -bv+jv+cv-kv;
    ACCESS2D(V, vx, vy++) = +bv-jv-dv+lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = +ev+iv-gv-kv;
    ACCESS2D(V, vx, vy++) = +fv+jv+gv+kv;
    ACCESS2D(V, vx, vy++) = -fv-jv+gv+kv;
    ACCESS2D(V, vx, vy++) = +fv+jv-hv-lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = -ev+iv+gv-kv;
    ACCESS2D(V, vx, vy++) = -fv+jv-gv+kv;
    ACCESS2D(V, vx, vy++) = +fv-jv-gv+kv;
    ACCESS2D(V, vx, vy++) = -fv+jv+hv-lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = +ev-mv-gv+ov;
    ACCESS2D(V, vx, vy++) = +fv-nv+gv-ov;
    ACCESS2D(V, vx, vy++) = -fv+nv+gv-ov;
    ACCESS2D(V, vx, vy++) = +fv-nv-hv+pv;

    __syncthreads();

    int toch = Tch % och;
    tch = Tch / och;

    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] = devU[((toch*ch+tch)*4+i)*4+j]*V[i][j]; 
    
    __syncthreads();

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            toch = Tch % och;
            // LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] += devfin[(((((tbs*p+tp)*q+tq)*ch+(tch+s))*och+toch)*4+i)*4+j];
        }
        __syncthreads();
    }

    if(tch == 0) 
    {

            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devfin[(((((tbs*p+tp)*q+tq)*ch+0)*och+toch)*4+i)*4+j];
    }
     __syncthreads();
  
}

__global__ void tile2(int bs, int p, int q, int ch, float *devin, float *devsum, float *devU, int h, int w, int och, float *devfin)
{
    int tbs, tp, tq, tch, tbsf, x;
    tbsf = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    x = threadIdx.x;
    tbs = tbsf%bs;

    int och_pb = MAX_THREAD/ch;
    int tf = tbsf / bs;
    int toch = x/ch + tf*(och_pb);
    tch = x%ch; 

    float V[4][4];
    
    // copy the tiles to thrtile
    // int offset1 = (tbs*ch + tch)*h*w;
    // for(int th = 2*tp, i = 0; i < 4; th++, i++)
    //     for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
    //         thrtile[i][j] = devin[offset1 + th*w + tw];

    int offset1 = (tbs*ch + tch)*h*w;
    float av, bv, cv, dv, ev, fv, gv, hv, iv, jv, kv, lv, mv, nv, ov, pv;
    int th = 2*tp, tw = 2*tq;
    av = ACCESS(devin, offset1, th*w + tw++);
    bv = ACCESS(devin, offset1, th*w + tw++);
    cv = ACCESS(devin, offset1, th*w + tw++);
    dv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    ev = ACCESS(devin, offset1, th*w + tw++);
    fv = ACCESS(devin, offset1, th*w + tw++);
    gv = ACCESS(devin, offset1, th*w + tw++);
    hv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    iv = ACCESS(devin, offset1, th*w + tw++);
    jv = ACCESS(devin, offset1, th*w + tw++);
    kv = ACCESS(devin, offset1, th*w + tw++);
    lv = ACCESS(devin, offset1, th*w + tw++);
    th++; tw = 2*tq;
    mv = ACCESS(devin, offset1, th*w + tw++);
    nv = ACCESS(devin, offset1, th*w + tw++);
    ov = ACCESS(devin, offset1, th*w + tw++);
    pv = ACCESS(devin, offset1, th*w + tw++);
    
    //Calculation of V
    int vx = 0, vy = 0;
    ACCESS2D(V, vx, vy++) = +av-iv-cv+kv;
    ACCESS2D(V, vx, vy++) = +bv-jv+cv-kv;
    ACCESS2D(V, vx, vy++) = -bv+jv+cv-kv;
    ACCESS2D(V, vx, vy++) = +bv-jv-dv+lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = +ev+iv-gv-kv;
    ACCESS2D(V, vx, vy++) = +fv+jv+gv+kv;
    ACCESS2D(V, vx, vy++) = -fv-jv+gv+kv;
    ACCESS2D(V, vx, vy++) = +fv+jv-hv-lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = -ev+iv+gv-kv;
    ACCESS2D(V, vx, vy++) = -fv+jv-gv+kv;
    ACCESS2D(V, vx, vy++) = +fv-jv-gv+kv;
    ACCESS2D(V, vx, vy++) = -fv+jv+hv-lv;
    vx++; vy = 0;
    ACCESS2D(V, vx, vy++) = +ev-mv-gv+ov;
    ACCESS2D(V, vx, vy++) = +fv-nv+gv-ov;
    ACCESS2D(V, vx, vy++) = -fv+nv+gv-ov;
    ACCESS2D(V, vx, vy++) = +fv-nv-hv+pv;
    
    __syncthreads();

    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] = devU[((toch*ch+tch)*4+i)*4+j]*V[i][j]; 
    
    __syncthreads();

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            //toch = Tch % och;
            // LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] += devfin[(((((tbs*p+tp)*q+tq)*ch+(tch+s))*och+toch)*4+i)*4+j];
        }
        __syncthreads();
    }

    if(tch == 0) 
    {

            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devfin[(((((tbs*p+tp)*q+tq)*ch+0)*och+toch)*4+i)*4+j];
    }
     __syncthreads();
  
}

__global__ void lastcal(int och, int p, int q, int bs, float *devsum, float *devY)
{
    int tbs, tp, tq, toch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    toch = threadIdx.x;

    int offset = (((tbs*och+toch)*p+tp)*q+tq)*16;
    float ay, by, cy, dy, ey, fy, gy, hy, iy, jy, ky, ly, my, ny, oy, py;
    int ind = 0;
    ay = ACCESS(devsum, offset, ind++);
    by = ACCESS(devsum, offset, ind++);
    cy = ACCESS(devsum, offset, ind++);
    dy = ACCESS(devsum, offset, ind++);
    ey = ACCESS(devsum, offset, ind++);
    fy = ACCESS(devsum, offset, ind++);
    gy = ACCESS(devsum, offset, ind++);
    hy = ACCESS(devsum, offset, ind++);
    iy = ACCESS(devsum, offset, ind++);
    jy = ACCESS(devsum, offset, ind++);
    ky = ACCESS(devsum, offset, ind++);
    ly = ACCESS(devsum, offset, ind++);
    my = ACCESS(devsum, offset, ind++);
    ny = ACCESS(devsum, offset, ind++);
    oy = ACCESS(devsum, offset, ind++);
    py = ACCESS(devsum, offset, ind++);
    
    ind = 0;
    offset = (((tbs*och+toch)*p+tp)*q+tq)*4;
    
    ACCESS(devY, offset, ind++) = ay+ey+iy+by+fy+jy+cy+gy+ky;
    ACCESS(devY, offset, ind++) = by+fy+jy-cy-gy-ky-dy-hy-ly;
    ACCESS(devY, offset, ind++) = ey-iy-my+fy-jy-ny+gy-ky-oy;
    ACCESS(devY, offset, ind++) = fy-jy-ny-gy+ky+oy-hy+ly+py;
}


float * WING::forward(int och, int ch, int bs, int h, int w, int pad, float *in, int &oph, int &opw, float *kwt, float& conv_time, float& overhead_time)
{
    conv_time = 0;
    overhead_time = 0;
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *devin, *devinnopad, *cutY, *devkwt, *devU;
    size_t insize = bs * ch * h * w * sizeof(float);
    int newh, neww;
 
    gpu_error(hipMalloc((void **) & devinnopad, insize));
    gpu_error(hipMemcpy(devinnopad, in, insize, hipMemcpyHostToDevice));

    newh = h + 2*pad;
    neww = w + 2*pad;
    oph = newh-2;
    opw = neww-2;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    insize = bs * ch * newh * neww * sizeof(float);
    gpu_error(hipMalloc((void **) & devin, insize));

    // call padding
    dim3 padgrid(bs, newh, neww);
    dim3 padblock(ch, 1, 1);
    
    hipEventRecord(start);
    paddev<<<padgrid,padblock>>>(devin, devinnopad, h, w, pad);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    overhead_time += milliseconds;

    gpu_error(hipFree(devinnopad));
    h = newh;
    w = neww;

    size_t kwtsize = och*ch*3*3*sizeof(float);    
    size_t usize = och*ch*4*4*sizeof(float);
    gpu_error(hipMalloc((void **) & devkwt, kwtsize));
    gpu_error(hipMalloc((void **) & devU, usize));
    gpu_error(hipMemcpy(devkwt, kwt, kwtsize, hipMemcpyHostToDevice));

    hipEventRecord(start);
    precompute<<<ch, och>>>(och, ch, devkwt, devU);
    hipEventRecord(stop);

    gpu_error(hipFree(devkwt));

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    overhead_time += milliseconds;

    size_t cutsize = bs*och*oph*opw*sizeof(float);
    cutY = (float *)malloc(cutsize);

    float *devsum, *devY, *devcutY;
    float *devfin;
    int p = max((h-2)/2, 0);
    int q = max((w-2)/2, 0);

    size_t finsize = MAX_B * p * q * ch * och * 4 * 4 * sizeof(float);
    size_t sumsize = bs * och * p * q * 4 * 4 * sizeof(float);
    size_t ysize = bs * och * p * q * 2 * 2 * sizeof(float);

 
    gpu_error(hipMalloc((void **) & devsum, sumsize));

    gpu_error(hipMalloc((void **) & devfin, finsize));

    size_t binsize = ch * newh * neww ;
    size_t dsumsize = och * p * q * 4 * 4 ;
    int bsg = (bs+MAX_B-1)/MAX_B;
    int prevb = 0;
    LOOP(bsg)
    {
        int currb = MAX_B;
        if(tbsg == bsg-1 && bs % MAX_B != 0)
            currb = bs % MAX_B;
        if(och*ch <= MAX_THREAD)
        {
            dim3 grid(currb, p, q); 
            dim3 block(och*ch, 1, 1);
            hipEventRecord(start);
            tile<<<grid, block>>>(currb, p, q, ch, devin + prevb*binsize, devsum + prevb*dsumsize, devU, h, w, och, devfin);
            hipEventRecord(stop);

            hipEventSynchronize(stop);
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            conv_time += milliseconds;

        }
        else
        {
            int f = (och*ch)/MAX_THREAD;
            dim3 grid(currb*f, p, q); 
            dim3 block(MAX_THREAD, 1, 1);
            hipEventRecord(start);
            tile2<<<grid, block>>>(currb, p, q, ch, devin + prevb*binsize, devsum + prevb*dsumsize, devU, h, w, och, devfin);   
            hipEventRecord(stop);

            hipEventSynchronize(stop);
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            conv_time += milliseconds;

        }
        prevb  += currb;
    }

    gpu_error(hipFree(devfin));
    gpu_error(hipFree(devin));    
    gpu_error(hipFree(devU));

    dim3 grid2(bs, p, q);
    dim3 block2(och, 1, 1);
    gpu_error(hipMalloc((void **) & devY, ysize));
    hipEventRecord(start);
    lastcal<<<grid2,block2>>>(och, p, q, bs, devsum, devY);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    overhead_time += milliseconds;

    gpu_error(hipFree(devsum));

    dim3 cutgrid(bs, p, q);
    dim3 cutblock(och,1,1);
    
    
    gpu_error(hipMalloc((void **) & devcutY, cutsize));
    hipEventRecord(start);
    cutpad<<<cutgrid, cutblock>>> (devY, devcutY, oph, opw);  
    hipEventRecord(stop);

 
    gpu_error(hipFree(devY));

    cudaSafeCall(hipMemcpy(cutY, devcutY, cutsize, hipMemcpyDeviceToHost));
    
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    overhead_time += milliseconds;

    gpu_error(hipFree(devcutY));

    gpu_error(hipEventDestroy(start));
    gpu_error(hipEventDestroy(stop));

    return cutY;

}
